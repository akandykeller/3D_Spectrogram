#include "hip/hip_runtime.h"
#include "spectrogram_kernel.cuh"
#include <hip/hip_vector_types.h>
#include <stdio.h>
#include <math.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime_api.h>

#include <SFML/Audio.hpp>

////////////////////////////////////////////////////////////////////////////////
// constants & defines
// Number of threads in a block.
#define BLOCK_SIZE 512
// Size of section of audio-sample which is analyzed at each timestep 
#define WINDOW_SIZE 1000.0
// Damping factor to use on amplitudes to avoid verticies leaving the visable space
#define DAMPING_FACTOR 0.000001

// macro for error-handling
#define gpuErrchk(ans) { gpuAssert((ans), (char*)__FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char* file, int line, bool abort=true)
{
  if (code != hipSuccess) 
  {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

#define WRAP(x,m) ((x)<(m)?(x):((x)-(m)))

// Flag for pingpong;
int pingpong = 0;

unsigned int numBodies;     // Number particles; determined at runtime.

////////////////////////////////////////////////////////////////////////////////
//! Window sample in place for gien time input
////////////////////////////////////////////////////////////////////////////////
__global__ void hannWindow(float4* newPos, Complex* sample, 
                            int size, int numBodies, float t, 
                            int row_size) {

    int i = WRAP(threadIdx.x + blockDim.x * blockIdx.x, size);

    while (i < size) {
        // First we also clear newPos for the next kernel.
        if (i < numBodies)
            newPos[i].y = 0.0;

        // Then apply the window function to our audio sample
        sample[i].x = (float) sample[i].x * 0.5 * 
                        (1.0 - cosf((float) (2.0 * M_PI * (i - t * size) / 
                            (float) ((float)size/ WINDOW_SIZE - 1.0))));
       
        // additionally use a hard cutoff to avoid some problems I was experiencing with the above
        if (i < t * size - ((float) size / WINDOW_SIZE) || i > t * size + ((float) size / WINDOW_SIZE))
            sample[i].x = 0;

        i += blockDim.x * gridDim.x;
    }

    syncthreads();
}

////////////////////////////////////////////////////////////////////////////////
//! Compute squared magnitudes in place
////////////////////////////////////////////////////////////////////////////////
__global__ void squareMagnitudes(float4* newPos, float4* oldPos, 
                                    Complex* d_signal, int sample_size, int numBodies, int row_size) {

    int i = WRAP(threadIdx.x + blockDim.x * blockIdx.x, sample_size);

    // Number of frequencied summed into each vertex
    int freqs_per_bucket = (int) floorf((float) sample_size / (float) row_size);
    int j;

    while (i < sample_size) {
        j = (int) floorf((float) i / (float) freqs_per_bucket);

        // Compute squared magnitude of transfored sample
        d_signal[i].x = d_signal[i].x * d_signal[i].x + d_signal[i].y * d_signal[i].y;
        
        // Add in the damped average value for the given vertex for
        // all applicable frequencies
        newPos[j].y += (d_signal[i].x / (float) freqs_per_bucket) * DAMPING_FACTOR;

        // Shift over old time values
        if (i >= row_size && i < numBodies + row_size) {
            newPos[i].y = oldPos[i - row_size].y;
        }

            // Damp the low end more
        if (i <= 75 || (i >= row_size - 75 && i <= row_size)) 
            newPos[i].y *= 0.0;


        i += blockDim.x * gridDim.x;
    }

    syncthreads();

    // Now map/chop the values so that they can be seen on the display
    i = WRAP(threadIdx.x + blockDim.x * blockIdx.x, sample_size);
    
    if (i < numBodies) {
        while (newPos[i].y > 25.0) {
            newPos[i].y = log(newPos[i].y);
        }
    }


}


////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runCuda(GLuint *vbo, Complex* complex_signal, int sample_size, float fraction_elapsed)
{
    // map OpenGL buffer object for writing from CUDA
    float4* oldPos;
    float4* newPos;

    unsigned int blocks = min((float)50, ceil(sample_size/(float)BLOCK_SIZE));

    // Map opengl buffers to CUDA.
    cudaGLMapBufferObject((void**)&oldPos, vbo[pingpong]);
    cudaGLMapBufferObject((void**)&newPos, vbo[!pingpong]);

    // Create space on gpu for the signal
    Complex *d_signal;
    gpuErrchk(hipMalloc((void **)&d_signal, sample_size * sizeof(Complex)));

    gpuErrchk(hipMemcpy((void *) d_signal, (void *) complex_signal, sample_size * sizeof(Complex),
                               hipMemcpyHostToDevice));

    /* First window the sample for the given time */
    hannWindow<<<blocks, BLOCK_SIZE>>>(newPos, d_signal, sample_size, numBodies, fraction_elapsed, (int) sqrt(numBodies));

    /* Compute fft on windowed sample */
    hipfftHandle plan;
    checkCudaErrors(hipfftPlan1d(&plan, sample_size, HIPFFT_C2C, 1));

    // Transform signal
    checkCudaErrors(hipfftExecC2C(plan, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, HIPFFT_FORWARD));

    // Set the y_pos of each vertex to the magnitude of the corresponding
    // value from the stft
    squareMagnitudes<<<blocks, BLOCK_SIZE>>>(newPos, oldPos, d_signal, sample_size, numBodies, sqrt(numBodies));

    int pos = ceil(25 * fraction_elapsed);
    printf("Progress: [+");
    for (int i = 0; i < 25; i++){
        if (i < pos)
            printf("=");
        else if (i == pos)
            printf(">");
        else
            printf(" ");
    }

    printf("]   %d %%\r", pos*4);

    // unmap buffer objects from cuda.
    cudaGLUnmapBufferObject(vbo[0]);
    cudaGLUnmapBufferObject(vbo[1]);

    //Destroy CUFFT context
    checkCudaErrors(hipfftDestroy(plan));

    gpuErrchk(hipFree(d_signal));

    //Switch buffers between old/new
    pingpong = !pingpong;
}


////////////////////////////////////////////////////////////////////////////////
//! Create VBO
////////////////////////////////////////////////////////////////////////////////
void createVBOs(GLuint* vbo)
{
    // create buffer object
    glGenBuffers(2, vbo);
    glBindBuffer(GL_ARRAY_BUFFER, vbo[0]);

    // initialize buffer object; this will be used as 'oldPos' initially
    unsigned int size = numBodies * 4 * sizeof(float);

    unsigned int plane_dim = ceil(sqrt(numBodies));

    float4* temppos = (float4*)malloc(size);

    for(int i = 0; i < numBodies; ++i)
    {
        temppos[i].x = ((i % plane_dim) - plane_dim / 2.) * 0.05;
        temppos[i].y = 0.;
        temppos[i].z = (floor(i / plane_dim) - plane_dim / 2.) * 0.05;
        temppos[i].w = 1.;
    }

    // Notice only vbo[0] has initial data!
    glBufferData(GL_ARRAY_BUFFER, size, temppos, GL_DYNAMIC_DRAW);

    free(temppos);

    // Create initial 'newPos' buffer
    glBindBuffer(GL_ARRAY_BUFFER, vbo[1]);
    glBufferData(GL_ARRAY_BUFFER, size, temppos, GL_DYNAMIC_DRAW);

    glBindBuffer(GL_ARRAY_BUFFER, 0);

    // register buffer objects with CUDA
    gpuErrchk(cudaGLRegisterBufferObject(vbo[0]));
    gpuErrchk(cudaGLRegisterBufferObject(vbo[1]));
}

////////////////////////////////////////////////////////////////////////////////
//! Delete VBO
////////////////////////////////////////////////////////////////////////////////
void deleteVBOs(GLuint* vbo)
{
    glBindBuffer(1, vbo[0]);
    glDeleteBuffers(1, &vbo[0]);
    glBindBuffer(1, vbo[1]);
    glDeleteBuffers(1, &vbo[1]);

    gpuErrchk(cudaGLUnregisterBufferObject(vbo[0]));
    gpuErrchk(cudaGLUnregisterBufferObject(vbo[1]));

    *vbo = 0;
}

////////////////////////////////////////////////////////////////////////////////
//! Returns the value of pingpong
////////////////////////////////////////////////////////////////////////////////
int getPingpong()
{
  return pingpong;
}

////////////////////////////////////////////////////////////////////////////////
//! Gets/sets the number of bodies
////////////////////////////////////////////////////////////////////////////////
int getNumBodies()
{
  return numBodies;
}

void setNumBodies(int n)
{
  numBodies = n;
}

